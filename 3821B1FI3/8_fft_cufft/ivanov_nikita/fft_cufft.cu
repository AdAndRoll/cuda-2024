// Copyright (c) 2024 Ivanov Nikita
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);
    std::vector<float> output(input.size(), 0.0f);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftComplex* d_input;
    hipfftComplex* d_output;
    hipMalloc((void**)&d_input, batch * n * sizeof(hipfftComplex));
    hipMalloc((void**)&d_output, batch * n * sizeof(hipfftComplex));

    hipMemcpy(d_input, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);

    hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);

    hipfftExecC2C(plan, d_output, d_input, HIPFFT_BACKWARD);

    float normalization_factor = 1.0f / n;
    hipfftComplex* d_normalized_output;
    hipMalloc((void**)&d_normalized_output, batch * n * sizeof(hipfftComplex));
    hipfftExecC2C(plan, d_input, d_normalized_output, HIPFFT_FORWARD);

    hipMemcpy(output.data(), d_normalized_output, output.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_normalized_output);

    hipfftDestroy(plan);

    return output;
}
