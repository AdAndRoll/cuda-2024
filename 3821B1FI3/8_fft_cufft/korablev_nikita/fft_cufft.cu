// Copyright (c) 2024 Korbalev Nikita
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);
    std::vector<float> output(input.size(), 0.0f);

    hipfftComplex* d_input;
    hipfftComplex* d_output;
    hipMalloc(&d_input, batch * n * sizeof(hipfftComplex));
    hipMalloc(&d_output, batch * n * sizeof(hipfftComplex));

    hipMemcpy(d_input, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_output, d_input, HIPFFT_BACKWARD);

    float normalizationFactor = 1.0f / n;
    hipfftComplex* d_normalized;
    hipMalloc(&d_normalized, batch * n * sizeof(hipfftComplex));
    hipfftExecC2C(plan, d_input, d_normalized, HIPFFT_FORWARD);

    hipMemcpy(output.data(), d_normalized, input.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_normalized);

    hipfftDestroy(plan);
    return output;
}
