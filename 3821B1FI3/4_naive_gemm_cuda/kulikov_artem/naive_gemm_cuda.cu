// Copyright (c) 2024 Kulikov Artem
#include <vector>
#include <cstdlib>

#include <hip/hip_runtime.h>


__global__ void myKernel(const float *a, const float *b,
                            float *const c, const size_t size) {
  size_t mIdx = blockIdx.y * blockDim.y + threadIdx.y;
  size_t kIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (mIdx < size && kIdx < size) {
    float *const c_ = &c[mIdx * size]; 
    float const cnst = a[mIdx * size + kIdx];
    const float *const b_ = &b[size * kIdx];

    for (int n = 0; n < size; n+=2) {
        c_[n] += cnst * b_[n];
        c_[n+1] += cnst * b_[n+1];
    }
  }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
    const std::vector<float>& b, int size) {
    std::vector<float> c(size * size);
    
    size_t sizeInBytes = size * size * sizeof(*a.data());
    
    float* d_a;
    hipMalloc(&d_a, sizeInBytes);
    float* d_b;
    hipMalloc(&d_b, sizeInBytes);
    float* d_c;
    hipMalloc(&d_c, sizeInBytes);
    
    hipMemcpy(d_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);
    
    const size_t sizeAxis = 64u;
    dim3 threadsPerBlock(
        sizeAxis,
        sizeAxis
    );
    dim3 numBlocks(
        (size + sizeAxis - 1) / sizeAxis,
        (size + sizeAxis - 1) / sizeAxis
    );
    
    myKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, size);

    hipMemcpy(c.data(), d_c, sizeInBytes, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return c;
}
