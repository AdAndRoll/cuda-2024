// Copyright (c) 2024 Ivanov Nikita
#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> c(n * n, 0.0f);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float* d_a;
    float* d_b;
    float* d_c;
    size_t sizeInBytes = n * n * sizeof(float);
    hipMalloc(&d_a, sizeInBytes);
    hipMalloc(&d_b, sizeInBytes);
    hipMalloc(&d_c, sizeInBytes);

    hipMemcpy(d_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

    hipMemcpy(c.data(), d_c, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipblasDestroy(handle);

    return c;
}
