#include <cstdlib>
#include <iostream>
#include <vector>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "gemm_cublas.h"


#include <chrono>

#define CUDA_CALL(callable) \
  { \
    auto error = callable; \
    if (error != hipSuccess) { \
      std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl; \
      std::exit(error); \
    } \
  }


#define CUBLAS_CALL(callable) \
  { \
    auto status = callable; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
      std::cerr << "cuBLAS error: " << status << std::endl; \
      std::exit(status); \
    } \
  }

std::vector<float> GemmUsingCUBLAS(const std::vector<float>& matrixA,
                                   const std::vector<float>& matrixB,
                                   int size) {
  hipDeviceProp_t deviceProperties{};
  CUDA_CALL(hipGetDeviceProperties(&deviceProperties, 0));

  size_t matrixSize = size * size;
  if (matrixA.size() != matrixSize || matrixB.size() != matrixSize) return {};

  std::vector<float> resultHost(matrixSize);
  size_t memorySize = matrixSize * sizeof(float);
  float alpha = 1.0f;
  float beta = 0.0f;

  float* deviceA = nullptr;
  float* deviceB = nullptr;
  float* deviceC = nullptr;

  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&deviceA), memorySize));
  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&deviceB), memorySize));
  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&deviceC), memorySize));

  CUDA_CALL(hipMemcpy(reinterpret_cast<void*>(deviceA), reinterpret_cast<const void*>(matrixA.data()), memorySize, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(reinterpret_cast<void*>(deviceB), reinterpret_cast<const void*>(matrixB.data()), memorySize, hipMemcpyHostToDevice));

  hipblasHandle_t cublasHandle{};
  CUBLAS_CALL(hipblasCreate(&cublasHandle));


  CUBLAS_CALL(hipblasSetMathMode(cublasHandle, HIPBLAS_TF32_TENSOR_OP_MATH));


  CUBLAS_CALL(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, deviceB, size, deviceA, size, &beta, deviceC, size));

  CUBLAS_CALL(hipblasDestroy(cublasHandle));

  CUDA_CALL(hipMemcpy(reinterpret_cast<void*>(resultHost.data()), reinterpret_cast<void*>(deviceC), memorySize, hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(reinterpret_cast<void*>(deviceA)));
  CUDA_CALL(hipFree(reinterpret_cast<void*>(deviceB)));
  CUDA_CALL(hipFree(reinterpret_cast<void*>(deviceC)));

  return resultHost;
}