#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "block_gemm_cuda.h"

std::vector<float> BlockGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int size) {
    
}