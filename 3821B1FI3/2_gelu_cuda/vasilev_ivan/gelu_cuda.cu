#include <iostream>
#include <vector>
#include <chrono>
#include <cstdlib>
#include <ctime>

#include <hip/hip_runtime.h>



const float mdsqrt2ipi = -1.59577f;   // - 2 * sqrt(2 / PI)
const float c = 0.044715f;


__global__ void geluKernel(const float* input, float* output, size_t size)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        const float x = input[i];
        //GELU: GELU(x) = 0.5 * x * (1 + tanh(sqrt(2 / pi) * (x + c * x^3)))
        float result = x * (0.5f * (1.0f + tanh(mdsqrt2ipi * (x + c * x * x * x))));
        output[i] = result;
    }
}


std::vector<float> GeluCUDA(const std::vector<float>& input) {
    const size_t size = input.size();
    std::vector<float> output(size);

    size_t sizeInBytes = size * sizeof(*input.data());

    float* d_input;
    float* d_output;
    

    hipMalloc(&d_input, sizeInBytes);
    hipMalloc(&d_output, sizeInBytes);
    

    hipMemcpy(d_input, input.data(), sizeInBytes, hipMemcpyHostToDevice);
    

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    

    geluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

    hipMemcpy(output.data(), d_output, sizeInBytes, hipMemcpyDeviceToHost);


    hipFree(d_input);
    hipFree(d_output);

    return output;
}