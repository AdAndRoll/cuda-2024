#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(error) \
    if (error != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CUBLAS_CHECK(status) \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error: " << status << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (a.size() != (n * n) || b.size() != (n * n)) return {};

    std::vector<float> c(n * n, 0.0f);
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    size_t bytes = n * n * sizeof(float);

    CUDA_CHECK(hipMalloc((void**)&d_a, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_b, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_c, bytes));

    CUDA_CHECK(hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_c, 0, bytes));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    const float alpha = 1.0f;
    const float beta = 0.0f;


    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n));

    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost));

    CUBLAS_CHECK(hipblasDestroy(handle));

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));

    return c;
}