#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(error) \
    if (error != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CUFFT_CHECK(status) \
    if (status != HIPFFT_SUCCESS) { \
        std::cerr << "cuFFT Error: " << status << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);

    std::vector<float> output(input.size(), 0.0f);

    hipfftComplex *d_input = nullptr, *d_output = nullptr;

    size_t bytes = input.size() * sizeof(float);

    CUDA_CHECK(hipMalloc((void**)&d_input, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_output, bytes));

    CUDA_CHECK(hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice));

    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlanMany(&plan, 1, &n, NULL, 1, n, NULL, 1, n, HIPFFT_C2C, batch));

    CUFFT_CHECK(hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD));

    CUFFT_CHECK(hipfftExecC2C(plan, d_output, d_input, HIPFFT_BACKWARD));

    CUFFT_CHECK(hipfftDestroy(plan));

    float inv_n = 1.0f / static_cast<float>(n);
    int total = n * batch;
    auto normalize_kernel = [] __global__ (hipfftComplex* data, int total, float inv_n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < total) {
            data[idx].x *= inv_n;
            data[idx].y *= inv_n;
        }
    };

    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    normalize_kernel<<<blocks, threads>>>(d_input, total, inv_n);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(output.data(), d_input, bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));

    return output;
}