#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "naive_gemm_cuda.h"

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

const int TILE_SIZE = 32;

__global__ void gemm_kernel_shared(const float* a, const float* b, float* c, int n) {
    __shared__ float tile_a[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_b[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;

    for (int k = 0; k < (n + TILE_SIZE - 1) / TILE_SIZE; ++k) {
        if (row < n && (k * TILE_SIZE + threadIdx.x) < n) {
            tile_a[threadIdx.y][threadIdx.x] = a[row * n + (k * TILE_SIZE + threadIdx.x)];
        }
        else {
            tile_a[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < n && (k * TILE_SIZE + threadIdx.y) < n) {
            tile_b[threadIdx.y][threadIdx.x] = b[(k * TILE_SIZE + threadIdx.y) * n + col];
        }
        else {
            tile_b[threadIdx.y][threadIdx.x] = 0.0f;
        }
        __syncthreads();
    
        for (int t = 0; t < TILE_SIZE; ++t) {
            sum += tile_a[threadIdx.y][t] * tile_b[t][threadIdx.x];
        }
      
    }

    if (row < n && col < n) {
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a, const std::vector<float>& b, int n) {
    size_t size = n * n * sizeof(float);

    float* d_a;
    float* d_b;
    float* d_c;

    checkCudaError(hipMalloc((void**)&d_a, size), "Failed to allocate memory for d_a");
    checkCudaError(hipMalloc((void**)&d_b, size), "Failed to allocate memory for d_b");
    checkCudaError(hipMalloc((void**)&d_c, size), "Failed to allocate memory for d_c");

    checkCudaError(hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice), "Failed to copy data for d_a");
    checkCudaError(hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice), "Failed to copy data for d_b");

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);

    gemm_kernel_shared << <numBlocks, threadsPerBlock >> > (d_a, d_b, d_c, n);
    checkCudaError(hipGetLastError(), "Error when starting the kernel");

    checkCudaError(hipDeviceSynchronize(), "Synchronization error");

    std::vector<float> c(n * n);
    checkCudaError(hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost), "Failed to copy data for c");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}