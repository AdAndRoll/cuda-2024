#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    size_t matrixSize = n * n * sizeof(float);

    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, matrixSize);
    hipMalloc((void**)&d_B, matrixSize);
    hipMalloc((void**)&d_C, matrixSize);

    hipMemcpy(d_A, a.data(), matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b.data(), matrixSize, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_T, HIPBLAS_OP_T,
                n, n, n,
                &alpha,
                d_B, n,
                d_A, n,
                &beta,
                d_C, n);

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_C, matrixSize, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return c;
}
