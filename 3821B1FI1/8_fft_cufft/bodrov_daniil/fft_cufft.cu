#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>

// Макрос для проверки ошибок CUDA
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        throw std::runtime_error(std::string("CUDA Error: ") + hipGetErrorString(err)); \
    }

// Макрос для проверки ошибок cuFFT
#define CUFFT_CHECK(err) \
    if (err != HIPFFT_SUCCESS) { \
        throw std::runtime_error(std::string("cuFFT Error: ") + cufftGetErrorString(err)); \
    }

// Функция для получения строкового описания ошибок cuFFT
const char* cufftGetErrorString(hipfftResult error) {
    switch(error) {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";
        default:
            return "Unknown cuFFT error";
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    // Проверка корректности размера входных данных
    if (input.size() % (2 * batch) != 0) {
        throw std::invalid_argument("Размер входного массива не соответствует формату (real, imaginary) для заданного batch.");
    }

    // Вычисление размера одного сигнала
    int n = input.size() / (2 * batch);

    // Размер данных в байтах
    size_t bytes = sizeof(float) * 2 * n * batch;

    // Указатели на устройства
    hipfftComplex *d_input = nullptr;
    hipfftComplex *d_forward = nullptr;
    hipfftComplex *d_inverse = nullptr;

    // Выделение памяти на устройстве
    CUDA_CHECK(hipMalloc((void**)&d_input, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_forward, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_inverse, bytes));

    // Копирование данных с хоста на устройство
    CUDA_CHECK(hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice));

    // Создание дескриптора cuFFT
    hipfftHandle plan_forward;
    hipfftHandle plan_inverse;

    // План для прямого FFT
    CUFFT_CHECK(hipfftPlan1d(&plan_forward, n, HIPFFT_C2C, batch));

    // План для обратного FFT
    CUFFT_CHECK(hipfftPlan1d(&plan_inverse, n, HIPFFT_C2C, batch));

    // Выполнение прямого FFT: d_input -> d_forward
    CUFFT_CHECK(hipfftExecC2C(plan_forward, d_input, d_forward, HIPFFT_FORWARD));

    // Выполнение обратного FFT: d_forward -> d_inverse
    CUFFT_CHECK(hipfftExecC2C(plan_inverse, d_forward, d_inverse, HIPFFT_BACKWARD));

    // Освобождение планов
    CUFFT_CHECK(hipfftDestroy(plan_forward));
    CUFFT_CHECK(hipfftDestroy(plan_inverse));

    // Выделение памяти для результата на хосте
    std::vector<float> output(2 * n * batch);

    // Копирование результата с устройства на хост
    CUDA_CHECK(hipMemcpy(output.data(), d_inverse, bytes, hipMemcpyDeviceToHost));

    // Освобождение памяти на устройстве
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_forward));
    CUDA_CHECK(hipFree(d_inverse));

    // Нормализация результата
    for(int i = 0; i < 2 * n * batch; ++i) {
        output[i] /= static_cast<float>(n);
    }

    return output;
}