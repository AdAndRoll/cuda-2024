#include "hip/hip_runtime.h"
// Copyright (c) 2024 Kashin Stepan

#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "gelu_cuda.h"

__global__ void ApplyGelu(const float* input, float* output, size_t length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < length) {
        constexpr float coeff1 = 1.595769122f;
        constexpr float coeff2 = 0.071354816f;

        float x = input[idx];
        output[idx] = x * (1 - 1 / (1.0f + __expf(x * (coeff1 + x * x * coeff2))));
    }
}

std::vector<float> ComputeGeluCUDA(const std::vector<float>& input) {
    if (input.empty()) return {};

    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, 0);

    size_t length = input.size();
    std::vector<float> output(length);

    size_t bytes = length * sizeof(float);
    int threads = deviceProps.maxThreadsPerBlock;
    int blocks = (length + threads - 1) / threads;

    float* devInput = nullptr;
    float* devOutput = nullptr;
    hipMalloc(&devInput, bytes);
    hipMalloc(&devOutput, bytes);

    hipMemcpy(devInput, input.data(), bytes, hipMemcpyHostToDevice);

    ApplyGelu<<<blocks, threads>>>(devInput, devOutput, length);

    hipDeviceSynchronize();
    hipMemcpy(output.data(), devOutput, bytes, hipMemcpyDeviceToHost);

    hipFree(devOutput);
    hipFree(devInput);

    return output;
}