#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

__global__ void GeluKernel(float *input, float *res, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  float x = input[i];
  res[i] = x * (1.0f -
                1.0f / (1.0f + __expf(x * fma(x * x, GELU_COEF2, GELU_COEF1))));
}

std::vector<float> GeluCUDA(const std::vector<float> &input) {
  auto size = input.size();
  std::vector<float> output(size);
  float *d_input, *d_output;

  hipMalloc(&d_input, input.size() * sizeof(float));
  hipMalloc(&d_output, output.size() * sizeof(float));
  hipMemcpy(d_input, input.data(), size * sizeof(float),
             hipMemcpyHostToDevice);

  int blockSize = 128;
  int numBlocks = (input.size() + blockSize - 1) / blockSize;

  GeluKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);

  hipDeviceSynchronize();
  hipMemcpy(output.data(), d_output, size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_output);
  hipFree(d_input);
  return output;
}
