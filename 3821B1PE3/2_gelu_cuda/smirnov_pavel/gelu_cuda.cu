#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

__global__ void geluKernel(const float* input, float* output, size_t size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    float x = input[i];
    float x3 = x * x * x;
    float arg = 2.0f / 3.14159265359f * (x + 0.044715f * x3);
    output[i] = 0.5f * x * (1.0f + tanh(arg));
  }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
  size_t size = input.size();
  std::vector<float> output(size);

  float* d_input;
  float* d_output;
  hipMalloc(&d_input, size * sizeof(float));
  hipMalloc(&d_output, size * sizeof(float));

  hipMemcpy(d_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
  geluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

  hipMemcpy(output.data(), d_output, size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);

  return output;
}

