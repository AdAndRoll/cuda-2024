#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <hip/hip_fp16.h>

#define BLOCK_SIZE 256

__global__ void geluKernel(const half* input, half* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        float x3 = x * x * x;
        float tanh_arg = sqrtf(2.0f / M_PI) * (x + 0.044715f * x3);
        output[idx] = 0.5f * x * (1.0f + tanhf(tanh_arg));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t size = input.size();
    std::vector<float> output(size);

    half* d_input;
    half* d_output;

    hipMalloc(&d_input, size * sizeof(half));
    hipMalloc(&d_output, size * sizeof(half));

    hipMemcpy(d_input, input.data(), size * sizeof(half), hipMemcpyHostToDevice);

    const int threadsPerBlock = BLOCK_SIZE;
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock; 
    geluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

    hipDeviceSynchronize(); 

    hipMemcpy(output.data(), d_output, size * sizeof(half), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}
